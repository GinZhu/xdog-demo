#include "hip/hip_runtime.h"
//
// by Jan Eric Kyprianidis <www.kyprianidis.com>
// Copyright (C) 2010-2012 Computer Graphics Systems Group at the
// Hasso-Plattner-Institut, Potsdam, Germany <www.hpi3d.de>
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
#include "gpu_st.h"
#include "gpu_gauss.h"


static texture<float, 2, hipReadModeElementType> texSRC1;
static texture<float4, 2, hipReadModeElementType> texSRC4;


template<typename T> __device__ T texSRC(float x, float y);
template<> inline __device__ float texSRC(float x, float y) { return tex2D(texSRC1, x, y); }
template<> inline __device__ float4 texSRC(float x, float y) { return tex2D(texSRC4, x, y); }


__global__ void imp_gray_st_central_diff( gpu_plm2<float4> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    float u = ( -tex2D(texSRC1, ix-1, iy  ) + tex2D(texSRC1, ix+1, iy  ) ) / 2;
    float v = ( -tex2D(texSRC1, ix,   iy-1) + tex2D(texSRC1, ix,   iy+1) ) / 2;
    float3 g = make_float3(u * u, v * v, u * v);

    dst(ix, iy) = make_float4( g, 1 );
}

                  
gpu_image<float4> gpu_st_central_diff( const gpu_image<float>& src ) {
    gpu_image<float4> dst(src.size());
    bind(&texSRC1, src);
    imp_gray_st_central_diff<<<src.blocks(), src.threads()>>>(dst);
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_gray_st_gaussian( gpu_plm2<float4> dst, float sigma ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    float twoSigma2 = 2.0f * sigma * sigma;
    int halfWidth = int(ceilf( 3.0f * sigma ));

    float u = 0;
    float v = 0;

    if (halfWidth > 0) {
        float sum = 0;
        for ( int i = -halfWidth; i <= halfWidth; ++i ) {
            for ( int j = -halfWidth; j <= halfWidth; ++j ) {
                float d = length(make_float2(i,j));
                float e = __expf( -d *d / twoSigma2 );
                float c = tex2D(texSRC1, ix + i, iy + j);
                u += i * e * c;
                v += j * e * c;
                sum += e;
            }
        }

        sum *= -sigma * sigma;
        u /= sum;
        v /= sum;
    }

    dst(ix, iy) = make_float4(u*u, v*v, u*v, 1);
}


__global__ void imp_color_st_gaussian( gpu_plm2<float4> dst, float sigma ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    float twoSigma2 = 2.0f * sigma * sigma;
    int halfWidth = int(ceilf( 3.0f * sigma ));

    float3 u = make_float3(0);
    float3 v = make_float3(0);

    if (halfWidth > 0) {
        float sum = 0;
        for ( int i = -halfWidth; i <= halfWidth; ++i ) {
            for ( int j = -halfWidth; j <= halfWidth; ++j ) {
                float d = length(make_float2(i,j));
                float e = __expf( -d *d / twoSigma2 );
                float3 c = make_float3(tex2D(texSRC4, ix + i, iy + j));
                u += i * e * c;
                v += j * e * c;
                sum += e;
            }
        }

        sum *= -sigma * sigma;
        u /= sum;
        v /= sum;
    }

    dst(ix, iy) = make_float4(dot(u, u), dot(v, v), dot(u, v), 1);
}

                                      
gpu_image<float4> gpu_st_gaussian( const gpu_image<float>& src, float sigma ) {
    gpu_image<float4> dst(src.size());
    bind(&texSRC1, src);
    imp_gray_st_gaussian<<<src.blocks(), src.threads()>>>(dst, sigma);
    GPU_CHECK_ERROR();
    return dst;
}


gpu_image<float4> gpu_st_gaussian( const gpu_image<float4>& src, float sigma ) {
    gpu_image<float4> dst(src.size());
    bind(&texSRC4, src);
    imp_color_st_gaussian<<<src.blocks(), src.threads()>>>(dst, sigma);
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_gray_st_sobel( gpu_plm2<float4> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    float u = (
           -1 * tex2D(texSRC1, ix-1, iy-1) +
           -2 * tex2D(texSRC1, ix-1, iy  ) + 
           -1 * tex2D(texSRC1, ix-1, iy+1) +
           +1 * tex2D(texSRC1, ix+1, iy-1) +
           +2 * tex2D(texSRC1, ix+1, iy  ) + 
           +1 * tex2D(texSRC1, ix+1, iy+1)
           ) / 8;

    float v = (
           -1 * tex2D(texSRC1, ix-1, iy-1) + 
           -2 * tex2D(texSRC1, ix,   iy-1) + 
           -1 * tex2D(texSRC1, ix+1, iy-1) +
           +1 * tex2D(texSRC1, ix-1, iy+1) +
           +2 * tex2D(texSRC1, ix,   iy+1) + 
           +1 * tex2D(texSRC1, ix+1, iy+1)
           ) / 8;
    
    dst(ix, iy) = make_float4(u*u, v*v, u*v, 1);
}


__global__ void imp_color_st_sobel( gpu_plm2<float4> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    float3 u = (
           -1 * make_float3(tex2D(texSRC4, ix-1, iy-1)) +
           -2 * make_float3(tex2D(texSRC4, ix-1, iy)) + 
           -1 * make_float3(tex2D(texSRC4, ix-1, iy+1)) +
           +1 * make_float3(tex2D(texSRC4, ix+1, iy-1)) +
           +2 * make_float3(tex2D(texSRC4, ix+1, iy)) + 
           +1 * make_float3(tex2D(texSRC4, ix+1, iy+1))
           ) / 8;

    float3 v = (
           -1 * make_float3(tex2D(texSRC4, ix-1, iy-1)) + 
           -2 * make_float3(tex2D(texSRC4, ix,   iy-1)) + 
           -1 * make_float3(tex2D(texSRC4, ix+1, iy-1)) +
           +1 * make_float3(tex2D(texSRC4, ix-1, iy+1)) +
           +2 * make_float3(tex2D(texSRC4, ix,   iy+1)) + 
           +1 * make_float3(tex2D(texSRC4, ix+1, iy+1))
           ) / 8;
    
    dst(ix, iy) = make_float4(dot(u, u), dot(v, v), dot(u, v), 1);
}


gpu_image<float4> gpu_st_sobel( const gpu_image<float>& src ) {
    gpu_image<float4> dst(src.size());
    bind(&texSRC1, src);
    imp_gray_st_sobel<<<src.blocks(), src.threads()>>>(dst);
    GPU_CHECK_ERROR();
    return dst;
}


gpu_image<float4> gpu_st_sobel( const gpu_image<float4>& src ) {
    gpu_image<float4> dst(src.size());
    bind(&texSRC4, src);
    imp_color_st_sobel<<<src.blocks(), src.threads()>>>(dst);
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_gray_st_scharr( gpu_plm2<float4> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    float u = (
          -0.183f * tex2D(texSRC1, ix-1, iy-1) +
          -0.634f * tex2D(texSRC1, ix-1, iy) + 
          -0.183f * tex2D(texSRC1, ix-1, iy+1) +
          +0.183f * tex2D(texSRC1, ix+1, iy-1) +
          +0.634f * tex2D(texSRC1, ix+1, iy) + 
          +0.183f * tex2D(texSRC1, ix+1, iy+1)
          ) * 0.5f;

    float v = (
          -0.183f * tex2D(texSRC1, ix-1, iy-1) + 
          -0.634f * tex2D(texSRC1, ix,   iy-1) + 
          -0.183f * tex2D(texSRC1, ix+1, iy-1) +
          +0.183f * tex2D(texSRC1, ix-1, iy+1) +
          +0.634f * tex2D(texSRC1, ix,   iy+1) + 
          +0.183f * tex2D(texSRC1, ix+1, iy+1)
          ) * 0.5f;
    
    float3 g = make_float3(u * u, v * v, u * v);
    dst(ix, iy) = make_float4( g, 1 );
}


__global__ void imp_color_st_scharr( gpu_plm2<float4> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    float3 u = (
          -0.183f * make_float3(tex2D(texSRC4, ix-1, iy-1)) +
          -0.634f * make_float3(tex2D(texSRC4, ix-1, iy)) + 
          -0.183f * make_float3(tex2D(texSRC4, ix-1, iy+1)) +
          +0.183f * make_float3(tex2D(texSRC4, ix+1, iy-1)) +
          +0.634f * make_float3(tex2D(texSRC4, ix+1, iy)) + 
          +0.183f * make_float3(tex2D(texSRC4, ix+1, iy+1))
          ) * 0.5f;

    float3 v = (
          -0.183f * make_float3(tex2D(texSRC4, ix-1, iy-1)) + 
          -0.634f * make_float3(tex2D(texSRC4, ix,   iy-1)) + 
          -0.183f * make_float3(tex2D(texSRC4, ix+1, iy-1)) +
          +0.183f * make_float3(tex2D(texSRC4, ix-1, iy+1)) +
          +0.634f * make_float3(tex2D(texSRC4, ix,   iy+1)) + 
          +0.183f * make_float3(tex2D(texSRC4, ix+1, iy+1))
          ) * 0.5f;

    dst(ix, iy) = make_float4( make_float3(dot(u,u), dot(v,v), dot(u,v)), 1 );
}


gpu_image<float4> gpu_st_scharr( const gpu_image<float>& src ) {
    gpu_image<float4> dst(src.size());
    bind(&texSRC1, src);
    imp_gray_st_scharr<<<src.blocks(), src.threads()>>>(dst);
    GPU_CHECK_ERROR();
    return dst;
}


gpu_image<float4> gpu_st_scharr( const gpu_image<float4>& src ) {
    gpu_image<float4> dst(src.size());
    bind(&texSRC4, src);
    imp_color_st_scharr<<<src.blocks(), src.threads()>>>(dst);
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_st_from_gradient( gpu_plm2<float4> dst, const gpu_plm2<float2> src ) {
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    float2 g = src(ix, iy);
    dst(ix,iy) = make_float4(g.x * g.x, g.y * g.y, g.x * g.y, 1);
}


gpu_image<float4> gpu_st_from_gradient( const gpu_image<float2>& src ) {
    gpu_image<float4> dst(src.size());
    imp_st_from_gradient<<<src.blocks(), src.threads()>>>(dst, src);
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_st_from_tangent( gpu_plm2<float4> dst, const gpu_plm2<float2> src ) {
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    float2 t = src(ix, iy);
    float2 g = make_float2(t.y, -t.x);
    dst(ix,iy) = make_float4(g.x * g.x, g.y * g.y, g.x * g.y, 1);
}


gpu_image<float4> gpu_st_from_tangent( const gpu_image<float2>& src ) {
    gpu_image<float4> dst(src.size());
    imp_st_from_tangent<<<src.blocks(), src.threads()>>>(dst, src);
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_st_tfm( gpu_plm2<float4> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    float4 g = tex2D(texSRC4, ix, iy);
    dst(ix, iy) = st2tfm(g);
}


gpu_image<float4> gpu_st_tfm( const gpu_image<float4>& st ) {
    gpu_image<float4> dst( st.size() );
    bind( &texSRC4, st );
    imp_st_tfm<<<dst.blocks(), dst.threads()>>>(dst); 
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_st_lfm( gpu_plm2<float4> dst, const gpu_plm2<float4> st ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    float4 g = st(ix, iy);
    dst(ix, iy) = st2lfm(g);
}


__global__ void imp_st_lfm( gpu_plm2<float4> dst, const gpu_plm2<float4> st, float alpha ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    float4 g = st(ix, iy);
    dst(ix, iy) = st2lfm(g, alpha);
}


gpu_image<float4> gpu_st_lfm( const gpu_image<float4>& st, float alpha ) {
    gpu_image<float4> dst( st.size() );
    if (alpha <= 0) 
        imp_st_lfm<<<dst.blocks(), dst.threads()>>>(dst, st); 
    else
        imp_st_lfm<<<dst.blocks(), dst.threads()>>>(dst, st, alpha); 
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_st_angle( gpu_plm2<float> dst, const gpu_plm2<float4> src ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;
    dst(ix, iy) = st2angle(src(ix, iy));
}


gpu_image<float> gpu_st_angle( const gpu_image<float4>& st ) {
    gpu_image<float> dst( st.size() );
    imp_st_angle<<<dst.blocks(), dst.threads()>>>(dst, st); 
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_st_anisotropy( gpu_plm2<float> dst, const gpu_plm2<float4> src ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;
    dst(ix, iy) = st2A(src(ix, iy));
}


gpu_image<float> gpu_st_anisotropy( const gpu_image<float4>& st ) {
    gpu_image<float> dst( st.size() );
    imp_st_anisotropy<<<dst.blocks(), dst.threads()>>>(dst, st); 
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_st_threshold_mag( gpu_plm2<float4> dst, float threshold2 ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if(ix >= dst.w || iy >= dst.h) 
        return;

    float3 g = make_float3(tex2D(texSRC4, ix, iy));
    float mag = g.x * g.x + g.y * g.y + 2 * g.z * g.z;
    if (mag < threshold2) {
        mag = 0;
    }
    dst(ix, iy) = make_float4(g, mag);
}

                                      
gpu_image<float4> gpu_st_threshold_mag( const gpu_image<float4>& st, float threshold ) {
    gpu_image<float4> dst( st.size() );
    bind( &texSRC4, st );
    imp_st_threshold_mag<<<dst.blocks(), dst.threads()>>>(dst, threshold*threshold); 
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_st_normalize( const gpu_plm2<float4> src, gpu_plm2<float4> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if(ix >= dst.w || iy >= dst.h) 
        return;

    float4 g = src(ix, iy);
    float mag = sqrtf(fmaxf(0, g.x * g.x + g.y * g.y + 2 * g.z * g.z));
    if (mag > 0) 
        dst(ix, iy) = make_float4(g.x / mag, g.y / mag, g.z / mag, 1);
    else
        dst(ix, iy) = make_float4(0);
}


gpu_image<float4> gpu_st_normalize( const gpu_image<float4>& st ) {
    gpu_image<float4> dst(st.size());
    imp_st_normalize<<<dst.blocks(), dst.threads()>>>( st, dst );
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_st_flatten( const gpu_plm2<float4> src, gpu_plm2<float4> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if(ix >= dst.w || iy >= dst.h) 
        return;

    float4 g = src(ix, iy);
    float phi = 0.5f * atan2(2 * g.z, g.x - g.y);
    float a = 0.5f * (g.y + g.x); 
    float b = 0.5f * sqrtf(fmaxf(0.0, g.y*g.y - 2*g.x*g.y + g.x*g.x + 4*g.z*g.z));
    float2 l = make_float2(a + b, a - b);

    float c = cosf(phi);
    float s = sinf(phi);

    dst(ix, iy) = make_float4(
        l.x*c*c,
        l.x*s*s,
        l.x*c*s,
        1
    );
}


gpu_image<float4> gpu_st_flatten( const gpu_image<float4>& st ) {
    gpu_image<float4> dst(st.size());
    imp_st_flatten<<<dst.blocks(), dst.threads()>>>( st, dst );
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_st_rotate( const gpu_plm2<float4> src, gpu_plm2<float4> dst, const float s, const float c ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if(ix >= dst.w || iy >= dst.h) 
        return;

    float4 g = src(ix, iy);
    dst(ix, iy) = make_float4(
        c*c*g.x + 2*s*c*g.z + s*s*g.y,
        s*s*g.x - 2*s*c*g.z + c*c*g.y,
        (c*c - s*s)*g.z + c*s*(g.y - g.x),
        1
    );
}


gpu_image<float4> gpu_st_rotate( const gpu_image<float4>& st, float angle ) {
    gpu_image<float4> dst(st.size());
    float phi = HIP_PI_F * angle / 180.0f;
    float s = sin(phi);
    float c = cos(phi);
    imp_st_rotate<<<dst.blocks(), dst.threads()>>>( st, dst, s, c );
    GPU_CHECK_ERROR();
    return dst;
}


