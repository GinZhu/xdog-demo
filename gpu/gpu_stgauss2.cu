#include "hip/hip_runtime.h"
//
// by Jan Eric Kyprianidis <www.kyprianidis.com>
// Copyright (C) 2010-2012 Computer Graphics Systems Group at the
// Hasso-Plattner-Institut, Potsdam, Germany <www.hpi3d.de>
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
#include "gpu_stgauss2.h"
#include "gpu_st.h"
#include "gpu_sampler.h"


static texture<float, 2, hipReadModeElementType> s_texSRC1;
static texture<float4, 2, hipReadModeElementType> s_texSRC4;

inline __host__ __device__ texture<float,2>& texSRC1() { return s_texSRC1; }
inline __host__ __device__ texture<float4,2>& texSRC4() { return s_texSRC4; }

static texture<float4, 2, hipReadModeElementType> s_texST;
inline __host__ __device__ texture<float4,2>& texST() { return s_texST; }


template <typename T, typename SRC>
struct stgauss2_filter {
     __device__ stgauss2_filter(const SRC& src, float sigma, float precision )
         : src_(src)
     { 
        radius_ = precision * sigma;
        twoSigma2_ = 2 * sigma * sigma;
        c_ = make_zero<T>();
        w_ = 0;
    }

    __device__ float radius() const {
        return radius_;
    }

    __device__ void operator()(float sign, float u, float2 p) {
        float k = __expf(-u * u / twoSigma2_);
        c_ += k * src_(p.x, p.y);
        w_ += k;
    }

    const SRC& src_;
    float radius_;
    float twoSigma2_;
    T c_;
    float w_;
};


template<int order, typename T, typename SRC, typename ST> 
__global__ void imp_stgauss2_filter( gpu_plm2<T> dst, SRC src, ST st, float sigma, float cos_max, 
                                     bool adaptive, float step_size, float precision ) 
{
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if(ix >= dst.w || iy >= dst.h) 
        return;

    float2 p0 = make_float2(ix + 0.5f, iy + 0.5f);
    if (adaptive) {
        float A = st2A(st(p0.x, p0.y));
        sigma *= 0.25f * (1 + A)*(1 + A);
    }
    stgauss2_filter<T,SRC> f(src, sigma, precision);
    if (order == 1) st_integrate_euler(p0, st, f, cos_max, dst.w, dst.h, step_size);
    if (order == 2) st_integrate_rk2(p0, st, f, cos_max, dst.w, dst.h, step_size);
    if (order == 4) st_integrate_rk4(p0, st, f, cos_max, dst.w, dst.h, step_size);
    dst(ix, iy) = f.c_ / f.w_;
}


gpu_image<float> gpu_stgauss2_filter( const gpu_image<float>& src, const gpu_image<float4>& st, 
                                      float sigma, float max_angle, bool adaptive,
                                      bool src_linear, bool st_linear, int order, float step_size,
                                      float precision )
{     
    if (sigma <= 0) return src;
    gpu_image<float> dst(src.size());

    gpu_sampler<float, texSRC1> src_sampler(src, src_linear? hipFilterModeLinear : hipFilterModePoint);
    float cos_max = cosf(radians(max_angle));

    if (src.size() == st.size()) {
        gpu_sampler<float4, texST> st_sampler(st, st_linear? hipFilterModeLinear : hipFilterModePoint);
        if (order == 1) imp_stgauss2_filter<1,float><<<dst.blocks(), dst.threads()>>>(dst, src_sampler, st_sampler, sigma, cos_max, adaptive, step_size, precision);
        else if (order == 2) imp_stgauss2_filter<2,float><<<dst.blocks(), dst.threads()>>>(dst, src_sampler, st_sampler, sigma, cos_max, adaptive, step_size, precision);
        else if (order == 4) imp_stgauss2_filter<4,float><<<dst.blocks(), dst.threads()>>>(dst, src_sampler, st_sampler, sigma, cos_max, adaptive, step_size, precision);
    } else {
        float2 s = make_float2((float)st.w() / src.w(), (float)st.h() / src.h());
        gpu_resampler<float4, texST> st_sampler(st, s, st_linear? hipFilterModeLinear : hipFilterModePoint);
        if (order == 1) imp_stgauss2_filter<1,float><<<dst.blocks(), dst.threads()>>>(dst, src_sampler, st_sampler, sigma, cos_max, adaptive, step_size, precision);
        else if (order == 2) imp_stgauss2_filter<2,float><<<dst.blocks(), dst.threads()>>>(dst, src_sampler, st_sampler, sigma, cos_max, adaptive, step_size, precision);
        else if (order == 4) imp_stgauss2_filter<4,float><<<dst.blocks(), dst.threads()>>>(dst, src_sampler, st_sampler, sigma, cos_max, adaptive, step_size, precision);
    }
    GPU_CHECK_ERROR();
    return dst;
}


gpu_image<float4> gpu_stgauss2_filter( const gpu_image<float4>& src, const gpu_image<float4>& st, 
                                       float sigma, float max_angle, bool adaptive,
                                       bool src_linear, bool st_linear, int order, float step_size,
                                       float precision )
{     
    if (sigma <= 0) return src;
    gpu_image<float4> dst(src.size());

    gpu_sampler<float4, texSRC4> src_sampler(src, src_linear? hipFilterModeLinear : hipFilterModePoint);
    float cos_max = cosf(radians(max_angle));

    if (src.size() == st.size()) {
        gpu_sampler<float4, texST> st_sampler(st, st_linear? hipFilterModeLinear : hipFilterModePoint);
        if (order == 1) imp_stgauss2_filter<1,float4><<<dst.blocks(), dst.threads()>>>(dst, src_sampler, st_sampler, sigma, cos_max, adaptive, step_size, precision);
        else if (order == 2) imp_stgauss2_filter<2,float4><<<dst.blocks(), dst.threads()>>>(dst, src_sampler, st_sampler, sigma, cos_max, adaptive, step_size, precision);
        else if (order == 4) imp_stgauss2_filter<4,float4><<<dst.blocks(), dst.threads()>>>(dst, src_sampler, st_sampler, sigma, cos_max, adaptive, step_size, precision);
    } else {
        float2 s = make_float2((float)st.w() / src.w(), (float)st.h() / src.h());
        gpu_resampler<float4, texST> st_sampler(st, s, st_linear? hipFilterModeLinear : hipFilterModePoint);
        if (order == 1) imp_stgauss2_filter<1,float4><<<dst.blocks(), dst.threads()>>>(dst, src_sampler, st_sampler, sigma, cos_max, adaptive, step_size, precision);
        else if (order == 2) imp_stgauss2_filter<2,float4><<<dst.blocks(), dst.threads()>>>(dst, src_sampler, st_sampler, sigma, cos_max, adaptive, step_size, precision);
        else if (order == 4) imp_stgauss2_filter<4,float4><<<dst.blocks(), dst.threads()>>>(dst, src_sampler, st_sampler, sigma, cos_max, adaptive, step_size, precision);
    }
    GPU_CHECK_ERROR();
    return dst;
}
