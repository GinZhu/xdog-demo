#include "hip/hip_runtime.h"
//
// by Jan Eric Kyprianidis <www.kyprianidis.com>
// Copyright (C) 2010-2012 Computer Graphics Systems Group at the
// Hasso-Plattner-Institut, Potsdam, Germany <www.hpi3d.de>
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
#include "gpu_util.h"


template <typename T>
__global__ void imp_adjust( const gpu_plm2<T> src, gpu_plm2<T> dst, T a, T b) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h)
        return;

    T c = src(ix, iy);
    dst(ix, iy) = a * c + b;
}                       


gpu_image<float> gpu_adjust( const gpu_image<float>& src, float a, float b ) {
    gpu_image<float> dst(src.size());
    imp_adjust<float><<<dst.blocks(), dst.threads()>>>(src, dst, a, b);
    GPU_CHECK_ERROR();
    return dst;
}


gpu_image<float4> gpu_adjust( const gpu_image<float4>& src, float4 a, float4 b ) {
    gpu_image<float4> dst(src.size());
    imp_adjust<float4><<<dst.blocks(), dst.threads()>>>(src, dst, a, b);
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_invert( const gpu_plm2<float> src, gpu_plm2<float> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h)
        return;

    float c = src(ix, iy);
    dst(ix, iy) = 1 - __saturatef(c);
}                       


gpu_image<float> gpu_invert( const gpu_image<float>& src ) {
    gpu_image<float> dst(src.size());
    imp_invert<<<dst.blocks(), dst.threads()>>>(src, dst);
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_invert( const gpu_plm2<float4> src, gpu_plm2<float4> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h)
        return;

    float4 c = src(ix, iy);
    dst(ix, iy) = make_float4( 1 - __saturatef(c.x), 
                               1 - __saturatef(c.y),
                               1 - __saturatef(c.z), 
                               1 );
}                       


gpu_image<float4> gpu_invert( const gpu_image<float4>& src ) {
    gpu_image<float4> dst(src.size());
    imp_invert<<<dst.blocks(), dst.threads()>>>(src, dst);
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_saturate( const gpu_plm2<float> src, gpu_plm2<float> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h)
        return;

    dst(ix, iy) = __saturatef(src(ix, iy));
}                       


gpu_image<float> gpu_saturate( const gpu_image<float>& src ) {
    gpu_image<float> dst(src.size());
    imp_saturate<<<dst.blocks(), dst.threads()>>>(src, dst);
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_saturate( const gpu_plm2<float4> src, gpu_plm2<float4> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h)
        return;

    float4 c = src(ix, iy);
    dst(ix, iy) = make_float4(__saturatef(c.x), __saturatef(c.y),__saturatef(c.z), 1);
}                       


gpu_image<float4> gpu_saturate( const gpu_image<float4>& src ) {
    gpu_image<float4> dst(src.size());
    imp_saturate<<<dst.blocks(), dst.threads()>>>(src, dst);
    GPU_CHECK_ERROR();
    return dst;
}


__global__ void imp_lerp( const gpu_plm2<float4> a, const gpu_plm2<float4> b, gpu_plm2<float4> dst, float t) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h)
        return;

    float4 ca = a(ix, iy);
    float4 cb = b(ix, iy);
    dst(ix, iy) = (1-t)*ca + t * cb;
}                       


gpu_image<float4> gpu_lerp( const gpu_image<float4>& a, const gpu_image<float4>& b, float t ) {
    assert(a.size() == b.size());
    gpu_image<float4> dst(a.size());
    imp_lerp<<<dst.blocks(), dst.threads()>>>( a, b, dst, t );
    GPU_CHECK_ERROR();
    return dst;
}
