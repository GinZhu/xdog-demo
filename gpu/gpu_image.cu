#include "hip/hip_runtime.h"
//
// by Jan Eric Kyprianidis <www.kyprianidis.com>
// Copyright (C) 2010-2012 Computer Graphics Systems Group at the
// Hasso-Plattner-Institut, Potsdam, Germany <www.hpi3d.de>
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
#include "gpu_image.h"


template <typename T>
__global__ void imp_set( gpu_plm2<T> dst, T value) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h) 
        return;

    dst(ix, iy) = value;
}


gpu_image<float> gpu_set( float value, unsigned w, unsigned h ) {
    gpu_image<float> dst(w,h);
    imp_set<float><<<dst.blocks(), dst.threads()>>>(  dst, value );
    GPU_CHECK_ERROR();
    return dst;
}


gpu_image<float4> gpu_set( float4 value, unsigned w, unsigned h ) {
    gpu_image<float4> dst(w,h);
    imp_set<float4><<<dst.blocks(), dst.threads()>>>(  dst, value );
    GPU_CHECK_ERROR();
    return dst;
}


template <typename T>
__global__ void imp_add( gpu_plm2<T> dst, const gpu_plm2<T> src0, const gpu_plm2<T> src1) {
    const unsigned ix = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned iy = blockDim.y * blockIdx.y + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h)
        return;

    dst(ix,iy) = src0(ix,iy) + src1(ix,iy);
}                       


gpu_image<float> gpu_add( const gpu_image<float>& src0, const gpu_image<float>& src1 ) {
    assert(src0.size() == src1.size());
    gpu_image<float> dst(src0.size());
    imp_add<float><<<dst.blocks(), dst.threads()>>>(dst, src0, src1);
    GPU_CHECK_ERROR();
    return dst;
}


gpu_image<float4> gpu_add( const gpu_image<float4>& src0, const gpu_image<float4>& src1 ) {
    assert(src0.size() == src1.size());
    gpu_image<float4> dst(src0.size());
    imp_add<float4><<<dst.blocks(), dst.threads()>>>(dst, src0, src1);
    GPU_CHECK_ERROR();
    return dst;
}


template <typename T>
__global__ void imp_mul( const gpu_plm2<T> src, gpu_plm2<T> dst, float value) {
    const unsigned ix = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned iy = blockDim.y * blockIdx.y + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h)
        return;

    T c = src(ix,iy);
    dst(ix,iy) = c * value;
}                       


gpu_image<float> gpu_mul( const gpu_image<float>& src, float value ) {
    gpu_image<float> dst(src.size());
    imp_mul<float><<<dst.blocks(), dst.threads()>>>(src, dst, value);
    GPU_CHECK_ERROR();
    return dst;
}


gpu_image<float4> gpu_mul( const gpu_image<float4>& src, float value ) {
    gpu_image<float4> dst(src.size());
    imp_mul<float4><<<dst.blocks(), dst.threads()>>>(src, dst, value);
    GPU_CHECK_ERROR();
    return dst;
}

